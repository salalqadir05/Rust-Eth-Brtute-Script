#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int count;
    hipError_t error = hipGetDeviceCount(&count);
    if (error != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(error));
        return 1;
    }
    printf("Found %d CUDA devices\n", count);
    return 0;
} 
