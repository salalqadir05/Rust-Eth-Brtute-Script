#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int count;
    hipError_t error = hipGetDeviceCount(&count);
    if (error != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(error));
        return 1;
    }
    printf("Found %d CUDA devices\n", count);
    
    // Get device properties
    for (int i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device %d: %s\n", i, prop.name);
        printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
        printf("  Total Memory: %lu MB\n", prop.totalGlobalMem / (1024*1024));
    }
    return 0;
} 
